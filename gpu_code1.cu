#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/functional.h>

using namespace std;

#define ni 24
#define nh 30
#define no 4
#define width 30
#define height 40

#define population_size 4096
#define natural_selection_rate 0.2
#define mutation_rate 0.01
#define generations 500

__global__ void myprint(float *nns, int size){
    int x = 5*(ni*nh+nh+nh*no+no);
    for(int i=x;i<x+24;i++)
        printf("%f ",nns[i]);
    printf("\n");
}

__global__ void initialise_nn(float *nns, unsigned int *random_int){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    nns[id] = (random_int[id] % 2) ? nns[id] : -nns[id];
}

__device__ void set_input(float input[], int x, int y, int fruitx, int fruity,
                    int tailx[], int taily[], int ntail){
    for(int i=0;i<ni;i++)
        input[i] = 0;

    // check up direction
	// check food
	if(fruitx == x && fruity < y)
        input[0] = 1;

    // check body
    for(int i=0;i<ntail;i++){
        if(tailx[i] == x && taily[i] < y){
            input[1] = 1;
            break;
        }
    }

    // check wall distance
    if(y != 0)
        input[2] = 1 / (float)y;

    // check down direction
    // check food
    if(fruitx == x && fruity > y)
        input[3] = 1;

    // check body
    for(int i=0;i<ntail;i++){
        if(tailx[i] == x && taily[i] > y){
            input[4] = 1;
            break;
        }
    }

    // check wall distance
    if(height-y != 0)
        input[5] = 1 / (float)(height-y);

    // check right direction
    // check food
    if(fruity == y && fruitx > x)
        input[6] = 1;

    // check body
    for(int i=0;i<ntail;i++){
        if(taily[i] == y && tailx[i] > x){
            input[7] = 1;
            break;
        }
    }

    // check wall distance
    if(width-x != 0)
        input[8] = 1 / (width-x);

    // check left direction
    // check food
    if(fruity == y && fruitx < x)
        input[9] = 1;

    // check body
    for(int i=0;i<ntail;i++){
        if(taily[i] == y && tailx[i] < x){
            input[10] = 1;
            break;
        }
    }

    // check wall distance
    if(x != 0)
        input[11] = 1 / (float)x;

    //check north-east direction
    int tempx = x, tempy = y;
    bool found_food = false, found_body = false;

    // check food and body
    while(tempx < width && tempy > 0){
        tempx++;
        tempy--;
        if(!found_food && tempx == fruitx && tempy == fruity){
            input[12] = 1;
            found_food = true;
        }
        if(!found_body){
            for(int i=0;i<ntail;i++){
                if(tempx == tailx[i] && tempy == taily[i]){
                    input[13] = 1;
                    found_body = true;
                    break;
                }
            }
        }
        if(found_body && found_food)
            break;
    }

    // check wall distance
    int min_value = min(width-x,y);
    float distance = sqrt(pow(min_value,2)*2);
    if(distance != 0)
        input[14] = 1 / distance; 

    //check north-west direction
    tempx = x, tempy = y;
    found_food = false, found_body = false;

    // check food and body
    while(tempx > 0 && tempy > 0){
        tempx--;
        tempy--;
        if(!found_food && tempx == fruitx && tempy == fruity){
            input[15] = 1;
            found_food = true;
        }
        if(!found_body){
            for(int i=0;i<ntail;i++){
                if(tempx == tailx[i] && tempy == taily[i]){
                    input[16] = 1;
                    found_body = true;
                    break;
                }
            }
        }
        if(found_body && found_food)
            break;
    }

    // check wall distance
    min_value = min(x,y);
    distance = sqrt(pow((min_value),2)*2);
    if(distance != 0)
        input[17] = 1 / distance; 

    //check south-west direction
    tempx = x, tempy = y;
    found_food = false, found_body = false;

    // check food and body
    while(tempx > 0 && tempy < height){
        tempx--;
        tempy++;
        if(!found_food && tempx == fruitx && tempy == fruity){
            input[18] = 1;
            found_food = true;
        }
        if(!found_body){
            for(int i=0;i<ntail;i++){
                if(tempx == tailx[i] && tempy == taily[i]){
                    input[19] = 1;
                    found_body = true;
                    break;
                }
            }
        }
        if(found_body && found_food)
            break;
    }

    // check wall distance
    min_value = min(x,height-y);
    distance = sqrt(pow((min_value),2)*2);
    if(distance != 0)
        input[20] = 1 / distance;

    //check south-east direction
    tempx = x, tempy = y;
    found_food = false, found_body = false;

    // check food and body
    while(tempx < width && tempy < height){
        tempx++;
        tempy++;
        if(!found_food && tempx == fruitx && tempy == fruity){
            input[21] = 1;
            found_food = true;
        }
        if(!found_body){
            for(int i=0;i<ntail;i++){
                if(tempx == tailx[i] && tempy == taily[i]){
                    input[22] = 1;
                    found_body = true;
                    break;
                }
            }
        }
        if(found_body && found_food)
            break;
    }

    // check wall distance
    min_value = min(width-x,height-y);
    distance = sqrt(pow((min_value),2)*2);
    if(distance != 0)
        input[23] = 1 / distance;
}

__device__ float forward(float input[], float weight[], float bias[], int len_i, int len_o, int index){
    float output = 0;
    for(int i=0;i<len_i;i++){
        output += weight[i*len_o+index] * input[i];
    }
    output += bias[index];

    // sigmoid function
    output = 1.0 / (1.0 + expf(-output));
    return output;
}

__global__ void play_game(float *nns, int *fitness, unsigned int *random_int_fruitx, unsigned int *random_int_fruity,
                        int parameter_size, int gen){

    // int id = blockIdx.x * blockDim.x + threadIdx.x;
    int snake_id = blockIdx.x;
    int parameter_id = threadIdx.x;

    extern __shared__ float nn[];
    // __shared__ float *nn;
    // nn = (float *)malloc(parameter_size*sizeof(float));    
    nn[parameter_id] = nns[snake_id*parameter_size+parameter_id];

    __syncthreads();

    float *w1 = &nn[0];
    float *b1 = &nn[ni*nh];
    float *w2 = &nn[ni*nh+nh];
    float *b2 = &nn[ni*nh+nh+nh*no];

    /* setup teh game */
    // STOP: 0, LEFT: 1, RIGHT: 2, UP: 3, DOWN: 4
    int dir = 0;

    // position of head
    int x = width/2;
    int y = height/2;

    // position of fruit
    int fruitx; 
    int fruity;
    int fruit_index = 0; 

    fruitx = random_int_fruitx[fruit_index] % width;
    fruity = random_int_fruity[fruit_index] % height;

    fruit_index++;

    //snake length
    int ntail = 2;

    int tailx[100], taily[100];

    int total_steps = 200;
	double total_reward = 0;
	double reward = 0;
	int steps = 0;
    __shared__ float input[ni];
    __shared__ float hidden_output[nh];
    __shared__ float output[no];
    
    while(true){
        set_input(input,x,y,fruitx,fruity,tailx,taily,ntail);
        
        if(parameter_id < nh){
            hidden_output[parameter_id] = forward(input,w1,b1,ni,nh,parameter_id);
        }

        __syncthreads();

        if(parameter_id < no){
            output[parameter_id] = forward(hidden_output,w2,b2,nh,no,parameter_id);
        }

        __syncthreads();

        if(parameter_id == 0){
            float max_value = output[0];
            float max_index = 0;
            for(int i=1;i<no;i++){
                if(output[i] > max_value){
                    max_value = output[i];
                    max_index = i;
                }
            }
            dir = max_index + 1;
        }

        __syncthreads();

        // if(snake_id == 0 && parameter_id == 0){
        //     for(int i=0;i<no;i++)
        //         printf("%f ",output[i]);
        //     printf("\n");
        //     // printf("%d\n",dir);
        // }

        int prevx = tailx[0];
        int prevy = taily[0];
        int prev2x, prev2y;
        tailx[0] = x;
        taily[0] = y;

        for(int i=1;i<ntail;i++)
        {
            prev2x = tailx[i];
            prev2y = taily[i];
            tailx[i] = prevx;
            taily[i] = prevy;
            prevx = prev2x;
            prevy = prev2y;
        }

        switch(dir)
        {
            case 1:
                x--;
                break;
            case 2:
                x++;
                break;
            case 3:
                y--;
                break;
            case 4:
                y++;
                break;
        }

        if(x >= width || x < 0 || y >= height || y < 0)
        {
            reward = -1;
        }

        for(int i =0; i<ntail;i++)
        {
            if(tailx[i]==x && taily[i]==y)
            {
                reward = -1;
            }
        }

        if(x==fruitx && y==fruity)
        {
            fruitx = random_int_fruitx[fruit_index] % width;
            fruity = random_int_fruity[fruit_index] % height;
            fruit_index++;
            ntail++;
            reward = 1;
        }

        if(reward == -1)
            break;

        total_reward += reward;
        reward = 0;

        steps += 1;

        if(reward > 0)
            total_steps = (total_steps+100 > 500) ? 500 : total_steps + 100;

        if(steps > total_steps)
            break;
    }
    
    __syncthreads();

    if(parameter_id == 0){
        fitness[snake_id] = total_reward;
        if(gen == 2 && snake_id == 5){
            fitness[snake_id] = 2;
        }
    }

    __syncthreads();

    // if(id == 0){
    //     for(int i=0;i<population_size;i++)
    //         printf("%d ",fitness[i]);
    //     printf("\n");
    // }
}

__global__ void select_top(float *nns, float *nns_new, int *indices){
    int id1 = blockIdx.x * blockDim.x + threadIdx.x;
    int id2 = indices[blockIdx.x] * blockDim.x + threadIdx.x;

    nns_new[id1] = nns[id2];
}

__global__ void myprint1(int *fitness, int *indices){
    for(int i=0;i<population_size;i++)
        printf("%d\t%d\n",fitness[i],indices[i]);
}

__global__ void intialise_indices(int *indices){
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    indices[id] = id;
}

__global__ void crossover(float *nns, unsigned int *random_int1, unsigned int *random_int2){
    int snake_id = blockIdx.x;
    int parameter_id = threadIdx.x;

    int top = population_size * natural_selection_rate;

    if(parameter_id <= random_int2[snake_id] % blockDim.x){
        nns[(top + snake_id) * blockDim.x + parameter_id] = nns[(random_int1[snake_id] % top) * blockDim.x + parameter_id];
    }
    else{
        nns[(top + snake_id) * blockDim.x + parameter_id] = nns[(random_int1[snake_id + blockDim.x] % top) * blockDim.x + parameter_id];
    }
}

__global__ void mutate(float *nns, float *random_float1, float *random_float2){
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if(random_float1[id] < mutation_rate){
        nns[id] += random_float2[id] / 5;
        if(nns[id] > 1)
            nns[id] = 1;
        if(nns[id] < -1)
            nns[id] = -1;
    }
}

int main(){
    srand(time(NULL));

    // number of parameters of neural network
    int parameter_size = ni*nh + nh + nh*no + no;
    cout<<"Parameter size: "<<parameter_size<<endl;

    // neural networks for device
    float *dnns, *dnns_new;

    // allocate memory for neural networks in device
    hipMalloc((void **)&dnns,population_size*parameter_size*sizeof(float));
    hipMalloc((void **)&dnns_new,population_size*parameter_size*sizeof(float));

    hiprandGenerator_t prng;
	
	// create pseudo random number generator
	hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_MT19937);
	hiprandSetPseudoRandomGeneratorSeed(prng, 42ULL);
	
	// initialise neural networks with uniform distribution
    hiprandGenerateUniform(prng, dnns, population_size*parameter_size);

    // create random number generator for integer values
    unsigned int *random_int;
    hipMalloc((void**) &random_int,population_size*parameter_size*sizeof(int));

	hiprandGenerate(prng,random_int,population_size*parameter_size); 
    
    // initialse the neural networks to have negative values also
    initialise_nn<<<population_size,parameter_size>>>(dnns,random_int);

    // myprint<<<1,1>>>(dnns,population_size*parameter_size);

    // hipDeviceSynchronize();

    int *dfitness, *dindices;

	// fitness score on host
	int *fitness = (int *) malloc(population_size*sizeof(int));

	// fitness score on device
	hipMalloc((void**) &dfitness,population_size*sizeof(int));
	hipMalloc((void**) &dindices,population_size*sizeof(int));

    thrust::device_ptr<int> fitness_ptr(dfitness);
    thrust::device_ptr<int> indices_ptr(dindices);

    unsigned int *random_int_fruitx;
    hipMalloc((void**) &random_int_fruitx,parameter_size*sizeof(int));
    unsigned int *random_int_fruity;
    hipMalloc((void**) &random_int_fruity,parameter_size*sizeof(int));

    unsigned int *random_int_crossover1;
    hipMalloc((void**) &random_int_crossover1,2*population_size*sizeof(int));
    unsigned int *random_int_crossover2;
    hipMalloc((void**) &random_int_crossover2,population_size*sizeof(int));

    float *random_float_mutate1;
    hipMalloc((void**) &random_float_mutate1,population_size*parameter_size*sizeof(float));
    float *random_float_mutate2;
    hipMalloc((void**) &random_float_mutate2,population_size*parameter_size*sizeof(float));
    
    int max_reward = 0;
    float avg_reward = 0;
    int global_max_reward = 0;
    int global_max_generation = 0;

    hipStream_t stream1;
    hipStreamCreate(&stream1);
    for(int k=0;k<generations;k++){
        // cout<<"Generation: "<<k+1<<endl;

        // intialise indices array corresponding to fitness array
        int num_threads = (population_size > 1024) ? 1024 : population_size;
        int num_blocks = population_size/1024 + 1;
        intialise_indices<<<num_blocks,num_threads>>>(dindices);

        hiprandSetStream(prng,stream1);

        // create random number generator for integer values of fruit
        hiprandGenerate(prng,random_int_fruitx,parameter_size);
        hiprandGenerate(prng,random_int_fruity,parameter_size);
        
        play_game<<<population_size,parameter_size,parameter_size*sizeof(float)>>>(dnns,dfitness,random_int_fruitx,random_int_fruity,parameter_size,k);
        
        hipMemcpy(fitness,dfitness,population_size*sizeof(int),hipMemcpyDeviceToHost);
        
        avg_reward = 0;
        max_reward = fitness[0];
        for(int i=1;i<population_size;i++){
            if(fitness[i] > max_reward){
                max_reward = fitness[i];
            }
            avg_reward += fitness[i];
        }
        avg_reward /= population_size;
        
        printf("generation: %d\tAverage fitness: %f\tMax reward: %d\n",k+1,avg_reward,max_reward);

        if(max_reward > global_max_reward){
            global_max_reward = max_reward;
            global_max_generation = k+1;
        }
        
        int top = population_size * natural_selection_rate;

        thrust::sort_by_key(fitness_ptr,fitness_ptr+population_size,indices_ptr,thrust::greater<int>());

        // myprint1<<<1,1>>>(dfitness,dindices);

        select_top<<<top,parameter_size>>>(dnns,dnns_new,dindices);

        float *temp = dnns_new;
        dnns_new = dnns;
        dnns = temp;

        hiprandGenerate(prng,random_int_crossover1,2*population_size);
        hiprandGenerate(prng,random_int_crossover2,population_size);

        hiprandGenerateUniform(prng,random_float_mutate1,population_size*parameter_size);
        hiprandGenerateNormal(prng,random_float_mutate2,population_size*parameter_size,0.0,1.0);

        hipStreamSynchronize(stream1);
        
        crossover<<<population_size-top,parameter_size>>>(dnns,random_int_crossover1,random_int_crossover2);

        mutate<<<population_size,parameter_size>>>(dnns,random_float_mutate1,random_float_mutate2);

        // myprint<<<1,1>>>(dnns,parameter_size);

        // hipDeviceSynchronize();
    }

    printf("Generation: %d\tGlobal max reward: %d\n",global_max_generation,global_max_reward);

    hipStreamDestroy(stream1);
    hipFree(dnns);
    hipFree(dnns_new);
    hipFree(random_int);
    hipFree(dfitness);
    hipFree(dindices);
    hipFree(random_int_fruitx);
    hipFree(random_int_fruity);
    hipFree(random_int_crossover1);
    hipFree(random_int_crossover2);
    hipFree(random_float_mutate1);
    hipFree(random_float_mutate2);
    free(fitness);

    return 0;
}