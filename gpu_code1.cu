#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <unistd.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/functional.h>

using namespace std;

#define ni 24
#define nh 30
#define no 4
#define width 30
#define height 20
#define max_snake_length 100

#define population_size 4096
#define natural_selection_rate 0.4
#define mutation_rate 0.05
#define generations 300

////////////////////////////////////////////////
//for color


#ifndef _COLORS_
#define _COLORS_

/* FOREGROUND */
#define RST  "\x1B[0m"
#define KRED  "\x1B[31m"
#define KGRN  "\x1B[32m"
#define KYEL  "\x1B[33m"
#define KBLU  "\x1B[34m"
#define KMAG  "\x1B[35m"
#define KCYN  "\x1B[36m"
#define KWHT  "\x1B[37m"

#define FRED(x) KRED x RST
#define FGRN(x) KGRN x RST
#define FYEL(x) KYEL x RST
#define FBLU(x) KBLU x RST
#define FMAG(x) KMAG x RST
#define FCYN(x) KCYN x RST
#define FWHT(x) KWHT x RST

#define BOLD(x) "\x1B[1m" x RST
#define UNDL(x) "\x1B[4m" x RST

#endif  /* _COLORS_ */

 

////////////////////////////////////////////////

__global__ void myprint(float *nns, int size){
    int x = 5*(ni*nh+nh+nh*no+no);
    for(int i=x;i<x+24;i++)
        printf("%f ",nns[i]);
    printf("\n");
}

__global__ void initialise_nn(float *nns, unsigned int *random_int){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    nns[id] = (random_int[id] % 2) ? nns[id] : -nns[id];
}

__device__ void set_input(float input[], int x, int y, int fruitx, int fruity,
                            int tailx[], int taily[], int ntail){
    for(int i=0;i<24;i++)
        input[i] = 1;
    // check up direction
    // check food
    if(fruitx == x && fruity < y){
        input[0] = y-fruity;
        input[0] /= height;
    }
    
    // check body
    for(int i=0;i<ntail;i++){
        if(tailx[i] == x && taily[i] < y){
            input[1] = y-taily[i];
            input[1] /= height;
            break;
        }
    }

    // check wall distance
    if(y != 0){
        input[2] = (float)y / height;
    }

    // check down direction
    // check food
    if(fruitx == x && fruity > y){
        input[3] = fruity - y;
        input[3] /= height;
    }
    
    // check body
    for(int i=0;i<ntail;i++){
        if(tailx[i] == x && taily[i] > y){
            input[4] = taily[i] - y;
            input[4] /= height;
            break;
        }
    }

    // check wall distance
    if(height-y != 0){
        input[5] = (float)(height-y) / height;
    }

    // check right direction
    // check food
    if(fruity == y && fruitx > x){
        input[6] = fruitx - x;
        input[6] /= height;
    }
    
    // check body
    for(int i=0;i<ntail;i++){
        if(taily[i] == y && tailx[i] > x){
            input[7] = tailx[i] -x;
            input[7] /= height;
            break;
        }
    }

    // check wall distance
    if(width-x != 0)
        input[8] = (float)(width-x) / height;

    // check left direction
    // check food
    if(fruity == y && fruitx < x){
        input[9] = x - fruitx;
        input[9] /= height;
    }
    
    // check body
    for(int i=0;i<ntail;i++){
        if(taily[i] == y && tailx[i] < x){
            input[10] = x - tailx[i];
            input[10] /= height;
            break;
        }
    }

    // check wall distance
    if(x != 0)
        input[11] = (float)x / height;

    //check north-east direction
    int tempx = x, tempy = y;
    bool found_food = false, found_body = false;
    int min_value = 0;
    float distance;

    // check food and body
    while(tempx < width && tempy > 0){
        tempx++;
        tempy--;
        if(!found_food && tempx == fruitx && tempy == fruity){
            min_value = min(fruitx,fruity);
            distance = sqrt(pow(min_value,2)*2);
            input[12] = distance / height;
            found_food = true;
        }
        if(!found_body){
            for(int i=0;i<ntail;i++){
                if(tempx == tailx[i] && tempy == taily[i]){
                    min_value = min(tailx[i],taily[i]);
                    distance = sqrt(pow(min_value,2)*2);
                    input[13] = distance / height;
                    found_body = true;
                    break;
                }
            }
        }
        if(found_body && found_food)
            break;
    }

    // check wall distance
    min_value = min(width-x,y);
    distance = sqrt(pow(min_value,2)*2);
    if(distance != 0)
        input[14] = distance / height; 

    //check north-west direction
    tempx = x, tempy = y;
    found_food = false, found_body = false;

    // check food and body
    while(tempx > 0 && tempy > 0){
        tempx--;
        tempy--;
        if(!found_food && tempx == fruitx && tempy == fruity){
            min_value = min(fruitx,fruity);
            distance = sqrt(pow(min_value,2)*2);
            input[15] = distance / height;
            found_food = true;
        }
        if(!found_body){
            for(int i=0;i<ntail;i++){
                if(tempx == tailx[i] && tempy == taily[i]){
                    min_value = min(tailx[i],taily[i]);
                    distance = sqrt(pow(min_value,2)*2);
                    input[16] = distance / height;
                    found_body = true;
                    break;
                }
            }
        }
        if(found_body && found_food)
            break;
    }

    // check wall distance
    min_value = min(x,y);
    distance = sqrt(pow((min_value),2)*2);
    if(distance != 0)
        input[17] = distance / height; 

    //check south-west direction
    tempx = x, tempy = y;
    found_food = false, found_body = false;

    // check food and body
    while(tempx > 0 && tempy < height){
        tempx--;
        tempy++;
        if(!found_food && tempx == fruitx && tempy == fruity){
            min_value = min(fruitx,fruity);
            distance = sqrt(pow(min_value,2)*2);
            input[18] = distance / height;
            found_food = true;
        }
        if(!found_body){
            for(int i=0;i<ntail;i++){
                if(tempx == tailx[i] && tempy == taily[i]){
                    min_value = min(tailx[i],taily[i]);
                    distance = sqrt(pow(min_value,2)*2);
                    input[19] = distance / height;
                    found_body = true;
                    break;
                }
            }
        }
        if(found_body && found_food)
            break;
    }

    // check wall distance
    min_value = min(x,height-y);
    distance = sqrt(pow((min_value),2)*2);
    if(distance != 0)
        input[20] = distance / height;

    //check south-east direction
    tempx = x, tempy = y;
    found_food = false, found_body = false;

    // check food and body
    while(tempx < width && tempy < height){
        tempx++;
        tempy++;
        if(!found_food && tempx == fruitx && tempy == fruity){
            min_value = min(fruitx,fruity);
            distance = sqrt(pow(min_value,2)*2);
            input[21] = distance / height;
            found_food = true;
        }
        if(!found_body){
            for(int i=0;i<ntail;i++){
                if(tempx == tailx[i] && tempy == taily[i]){
                    min_value = min(tailx[i],taily[i]);
                    distance = sqrt(pow(min_value,2)*2);
                    input[22] = distance / height;
                    found_body = true;
                    break;
                }
            }
        }
        if(found_body && found_food)
            break;
    }

    // check wall distance
    min_value = min(width-x,height-y);
    distance = sqrt(pow((min_value),2)*2);
    if(distance != 0)
        input[23] = distance / height;
}

__device__ float forward(float input[], float weight[], float bias[], int len_i, int len_o, int index){
    float output = 0;
    for(int i=0;i<len_i;i++){
        output += weight[i*len_o+index] * input[i];
    }
    output += bias[index];

    // sigmoid function
    output = 1.0 / (1.0 + expf(-output));
    return output;
}

__global__ void play_game(float *nns, int *fitness, unsigned int *random_int_fruitx, unsigned int *random_int_fruity,
                        int parameter_size){

    // int id = blockIdx.x * blockDim.x + threadIdx.x;
    int snake_id = blockIdx.x;
    int parameter_id = threadIdx.x;

    extern __shared__ float nn[];
    // __shared__ float *nn;
    // nn = (float *)malloc(parameter_size*sizeof(float));    
    nn[parameter_id] = nns[snake_id*parameter_size+parameter_id];

    __syncthreads();

    float *w1 = &nn[0];
    float *b1 = &nn[ni*nh];
    float *w2 = &nn[ni*nh+nh];
    float *b2 = &nn[ni*nh+nh+nh*no];

    /* setup teh game */
    // STOP: 0, LEFT: 1, RIGHT: 2, UP: 3, DOWN: 4
    int dir = 0;

    // position of head
    int x = width/2;
    int y = height/2;

    // position of fruit
    int fruitx; 
    int fruity;
    int fruit_index = snake_id * max_snake_length; 

    fruitx = random_int_fruitx[fruit_index] % width;
    fruity = random_int_fruity[fruit_index] % height;

    fruit_index++;

    //snake length
    int ntail = 3;

    int tailx[max_snake_length], taily[max_snake_length];

    int total_steps = 200;
	double total_reward = 0;
	double reward = 0;
	int steps = 0;
    __shared__ float input[ni];
    __shared__ float hidden_output[nh];
    __shared__ float output[no];
    
    while(true){
        set_input(input,x,y,fruitx,fruity,tailx,taily,ntail);
        
        if(parameter_id < nh){
            hidden_output[parameter_id] = forward(input,w1,b1,ni,nh,parameter_id);
        }

        __syncthreads();

        if(parameter_id < no){
            output[parameter_id] = forward(hidden_output,w2,b2,nh,no,parameter_id);
        }

        __syncthreads();

        if(parameter_id == 0){
            float max_value = output[0];
            float max_index = 0;
            for(int i=1;i<no;i++){
                if(output[i] > max_value){
                    max_value = output[i];
                    max_index = i;
                }
            }
            dir = max_index + 1;
        }

        __syncthreads();

        // if(snake_id == 0 && parameter_id == 0){
        //     for(int i=0;i<no;i++)
        //         printf("%f ",output[i]);
        //     printf("\n");
        //     // printf("%d\n",dir);
        // }

        int prevx = tailx[0];
        int prevy = taily[0];
        int prev2x, prev2y;
        tailx[0] = x;
        taily[0] = y;

        for(int i=1;i<ntail;i++)
        {
            prev2x = tailx[i];
            prev2y = taily[i];
            tailx[i] = prevx;
            taily[i] = prevy;
            prevx = prev2x;
            prevy = prev2y;
        }

        switch(dir)
        {
            case 1:
                x--;
                break;
            case 2:
                x++;
                break;
            case 3:
                y--;
                break;
            case 4:
                y++;
                break;
        }

        if(x >= width || x < 0 || y >= height || y < 0)
        {
            reward = -1;
        }

        for(int i =0; i<ntail;i++)
        {
            if(tailx[i]==x && taily[i]==y)
            {
                reward = -1;
            }
        }

        if(x==fruitx && y==fruity)
        {
            fruitx = random_int_fruitx[fruit_index] % width;
            fruity = random_int_fruity[fruit_index] % height;
            fruit_index++;
            ntail++;
            reward = 1;
        }

        if(reward == -1)
            break;

        total_reward += reward;
        reward = 0;

        steps += 1;

        if(reward > 0)
            total_steps = (total_steps+100 > 500) ? 500 : total_steps + 100;

        if(steps > total_steps)
            break;
    }
    
    __syncthreads();

    if(parameter_id == 0){
        fitness[snake_id] = total_reward;
        // if(gen == 2 && snake_id == 5){
        //     fitness[snake_id] = 2;
        // }
    }

    // __syncthreads();

    // if(id == 0){
    //     for(int i=0;i<population_size;i++)
    //         printf("%d ",fitness[i]);
    //     printf("\n");
    // }
}

__global__ void select_top(float *nns, float *nns_new, int *indices){
    int id1 = blockIdx.x * blockDim.x + threadIdx.x;
    int id2 = indices[blockIdx.x] * blockDim.x + threadIdx.x;

    nns_new[id1] = nns[id2];
}

__global__ void myprint1(int *fitness, int *indices){
    for(int i=0;i<population_size;i++)
        printf("%d\t%d\n",fitness[i],indices[i]);
}

__global__ void intialise_indices(int *indices){
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    indices[id] = id;
}

__global__ void crossover(float *nns, unsigned int *random_int1, unsigned int *random_int2){
    int snake_id = blockIdx.x;
    int parameter_id = threadIdx.x;

    int top = population_size * natural_selection_rate;

    if(parameter_id <= random_int2[snake_id] % blockDim.x){
        nns[(top + snake_id) * blockDim.x + parameter_id] = nns[(random_int1[snake_id] % top) * blockDim.x + parameter_id];
    }
    else{
        nns[(top + snake_id) * blockDim.x + parameter_id] = nns[(random_int1[snake_id + blockDim.x] % top) * blockDim.x + parameter_id];
    }
}

__global__ void mutate(float *nns, float *random_float1, float *random_float2){
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if(random_float1[id] < mutation_rate){
        nns[id] += random_float2[id] / 5;
        if(nns[id] > 1)
            nns[id] = 1;
        if(nns[id] < -1)
            nns[id] = -1;
    }
}

// game variables
bool gameover;
int dir;
int score;
int x, y;
int fruitx, fruity;
int tailx[max_snake_length], taily[max_snake_length];
int ntail;

void setup(){
    gameover=false;
    dir = 0;
    x = width/2;
    y = height/2;
    fruitx=rand() % width;
    fruity=rand() % height;
    ntail = 3;
    score = 0;
}

void draw(){

    system("clear");

    for(int i =0; i<width+2;i++)
        cout << FGRN("+"); 

    cout<<endl;

    for(int i=0;i<height;i++)
    {
        for(int j=0; j<width;j++)
        {
            if(j == 0)
                cout << FBLU("+"); 

            if(i==y && j==x)
            {
                cout << FGRN("0");
            }

            else if(i == fruity && j == fruitx)
            {
                cout << FGRN("*"); 
            }
            
            else
            {
                bool print = false;
                for(int k =0;k < ntail;k++)
                {
                    if(tailx[k] == j && taily[k] ==i)
                    {
                        cout<<FWHT("o"); 
                        print = true;
                    }
                }
                if(!print)
                {
                    cout<<" ";
                }

            }

            if(j==width-1)
                cout << FRED("+");
        }

        cout<<endl;
    }
    for(int i = 0;i<width+2;i++)
    cout << FBLU("+");

    cout<<endl;

    cout<< UNDL(FRED("Score:")) <<score<<"\t"<<endl; 

    cout<< FMAG("hi");
    cout<<x<<" "<<y<<" : "<<fruitx<<" "<<fruity<<" : "<<tailx[ntail-1]<<" "<<taily[ntail-1]<<endl;
 
}

void set_input1(float *input){
	for(int i=0;i<24;i++)
		input[i] = 1;
	// check up direction
	// check food
	if(fruitx == x && fruity < y){
		input[0] = y-fruity;
		input[0] /= height;
	}
	
	// check body
	for(int i=0;i<ntail;i++){
		if(tailx[i] == x && taily[i] < y){
			input[1] = y-taily[i];
			input[1] /= height;
			break;
		}
	}

	// check wall distance
	if(y != 0){
		input[2] = (float)y / height;
	}

	// check down direction
	// check food
	if(fruitx == x && fruity > y){
		input[3] = fruity - y;
		input[3] /= height;
	}
	
	// check body
	for(int i=0;i<ntail;i++){
		if(tailx[i] == x && taily[i] > y){
			input[4] = taily[i] - y;
			input[4] /= height;
			break;
		}
	}

	// check wall distance
	if(height-y != 0){
		input[5] = (float)(height-y) / height;
	}

	// check right direction
	// check food
	if(fruity == y && fruitx > x){
		input[6] = fruitx - x;
		input[6] /= height;
	}
	
	// check body
	for(int i=0;i<ntail;i++){
		if(taily[i] == y && tailx[i] > x){
			input[7] = tailx[i] -x;
			input[7] /= height;
			break;
		}
	}

	// check wall distance
	if(width-x != 0)
		input[8] = (float)(width-x) / height;

	// check left direction
	// check food
	if(fruity == y && fruitx < x){
		input[9] = x - fruitx;
		input[9] /= height;
	}
	
	// check body
	for(int i=0;i<ntail;i++){
		if(taily[i] == y && tailx[i] < x){
			input[10] = x - tailx[i];
			input[10] /= height;
			break;
		}
	}

	// check wall distance
	if(x != 0)
		input[11] = (float)x / height;

	//check north-east direction
	int tempx = x, tempy = y;
	bool found_food = false, found_body = false;
	int min_value = 0;
	float distance;

	// check food and body
	while(tempx < width && tempy > 0){
		tempx++;
		tempy--;
		if(!found_food && tempx == fruitx && tempy == fruity){
			min_value = min(fruitx,fruity);
			distance = sqrt(pow(min_value,2)*2);
			input[12] = distance / height;
			found_food = true;
		}
		if(!found_body){
			for(int i=0;i<ntail;i++){
				if(tempx == tailx[i] && tempy == taily[i]){
					min_value = min(tailx[i],taily[i]);
					distance = sqrt(pow(min_value,2)*2);
					input[13] = distance / height;
					found_body = true;
					break;
				}
			}
		}
		if(found_body && found_food)
			break;
	}

	// check wall distance
	min_value = min(width-x,y);
	distance = sqrt(pow(min_value,2)*2);
	if(distance != 0)
		input[14] = distance / height; 

	//check north-west direction
	tempx = x, tempy = y;
	found_food = false, found_body = false;

	// check food and body
	while(tempx > 0 && tempy > 0){
		tempx--;
		tempy--;
		if(!found_food && tempx == fruitx && tempy == fruity){
			min_value = min(fruitx,fruity);
			distance = sqrt(pow(min_value,2)*2);
			input[15] = distance / height;
			found_food = true;
		}
		if(!found_body){
			for(int i=0;i<ntail;i++){
				if(tempx == tailx[i] && tempy == taily[i]){
					min_value = min(tailx[i],taily[i]);
					distance = sqrt(pow(min_value,2)*2);
					input[16] = distance / height;
					found_body = true;
					break;
				}
			}
		}
		if(found_body && found_food)
			break;
	}

	// check wall distance
	min_value = min(x,y);
	distance = sqrt(pow((min_value),2)*2);
	if(distance != 0)
		input[17] = distance / height; 

	//check south-west direction
	tempx = x, tempy = y;
	found_food = false, found_body = false;

	// check food and body
	while(tempx > 0 && tempy < height){
		tempx--;
		tempy++;
		if(!found_food && tempx == fruitx && tempy == fruity){
			min_value = min(fruitx,fruity);
			distance = sqrt(pow(min_value,2)*2);
			input[18] = distance / height;
			found_food = true;
		}
		if(!found_body){
			for(int i=0;i<ntail;i++){
				if(tempx == tailx[i] && tempy == taily[i]){
					min_value = min(tailx[i],taily[i]);
					distance = sqrt(pow(min_value,2)*2);
					input[19] = distance / height;
					found_body = true;
					break;
				}
			}
		}
		if(found_body && found_food)
			break;
	}

	// check wall distance
	min_value = min(x,height-y);
	distance = sqrt(pow((min_value),2)*2);
	if(distance != 0)
		input[20] = distance / height;

	//check south-east direction
	tempx = x, tempy = y;
	found_food = false, found_body = false;

	// check food and body
	while(tempx < width && tempy < height){
		tempx++;
		tempy++;
		if(!found_food && tempx == fruitx && tempy == fruity){
			min_value = min(fruitx,fruity);
			distance = sqrt(pow(min_value,2)*2);
			input[21] = distance / height;
			found_food = true;
		}
		if(!found_body){
			for(int i=0;i<ntail;i++){
				if(tempx == tailx[i] && tempy == taily[i]){
					min_value = min(tailx[i],taily[i]);
					distance = sqrt(pow(min_value,2)*2);
					input[22] = distance / height;
					found_body = true;
					break;
				}
			}
		}
		if(found_body && found_food)
			break;
	}

	// check wall distance
	min_value = min(width-x,height-y);
	distance = sqrt(pow((min_value),2)*2);
	if(distance != 0)
		input[23] = distance / height;
}

void forward(float *input, float *output, float*w1,
            float *w2, float *b1, float *b2){
    float *layer1 = (float *)malloc(nh*sizeof(float));
    for(int i=0;i<nh;i++){
        layer1[i] = 0;
        for(int j=0;j<ni;j++){
            layer1[i] += input[j]*w1[j*nh+i];
        }
        layer1[i] += b1[i];

        // sigmoid activation
        layer1[i] = 1 / (1 + exp(-layer1[i]));
    }

    // for(int i=0;i<nh;i++)
    //     cout<<layer1[i]<<" ";
    // cout<<endl;

    for(int i=0;i<no;i++){
        output[i] = 0;
        for(int j=0;j<nh;j++){
            output[i] += layer1[j]*w2[j*no+i];
        }
        output[i] += b2[i];

        // sigmoid activation
        output[i] = 1 / (1 + exp(-output[i]));
    }

    // for(int i=0;i<no;i++)  
    //     cout<<output[i]<<" ";
    // cout<<endl;

    free(layer1);
}

void get_direction(float *parameters){
    float *w1 = &parameters[0];
    float *b1 = &parameters[ni*nh];
    float *w2 = &parameters[ni*nh+nh];
    float *b2 = &parameters[ni*nh+nh+nh*no];

    float *input = (float *)malloc(ni*sizeof(float));
    set_input1(input);

    float *output = (float *)malloc(no*sizeof(float));

    forward(input,output,w1,w2,b1,b2);

    // for(int i=0;i<no;i++)
    //     cout<<output[i]<<" ";
    // cout<<endl;

    int index = -1;
    float max = INT16_MIN;
    for(int j=0;j<no;j++){
        if(output[j] > max){
            max = output[j];
            index = j;
        }
    }

    dir = index + 1;

    free(input);
    free(output);
}

void logic(){

    int prevx = tailx[0];
    int prevy = taily[0];
    int prev2x, prev2y;
    tailx[0] = x;
    taily[0] = y;

    for(int i=1;i<ntail;i++)
    {
        prev2x = tailx[i];
        prev2y = taily[i];
        tailx[i] = prevx;
        taily[i] = prevy;
        prevx = prev2x;
        prevy = prev2y;
    }

    switch(dir)
    {
        case 1:
            x--;
            break;
        case 2:
            x++;
            break;
        case 3:
            y--;
            break;
        case 4:
            y++;
            break;
    }

    if(x >= width || x < 0 || y >= height || y < 0)
    {
        gameover=true;
        cout<<"GAME OVER"<<endl;
    }

    for(int i =0; i<ntail;i++)
    {
        if(tailx[i]==x && taily[i]==y)
        {
            gameover = true;
            cout<<"GAME OVER"<<endl;
        }
    }

    if(x==fruitx && y==fruity)
    {
        score = score +1000;
        fruitx=rand() % width;
        fruity=rand() % height;
        ntail++;
    }
}

void view_game(float *parameters){
    setup();
    while(!gameover)
    {
        draw();
        get_direction(parameters);
        logic();
        usleep(150000);
    }
}

int main(){
    srand(time(NULL));

    // number of parameters of neural network
    int parameter_size = ni*nh + nh + nh*no + no;
    cout<<"Parameter size: "<<parameter_size<<endl;

    // neural networks for device
    float *dnns, *dnns_new;

    // allocate memory for neural networks in device
    hipMalloc((void **)&dnns,population_size*parameter_size*sizeof(float));
    hipMalloc((void **)&dnns_new,population_size*parameter_size*sizeof(float));

    hiprandGenerator_t prng;
	
	// create pseudo random number generator
	hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_MT19937);
	hiprandSetPseudoRandomGeneratorSeed(prng, 42ULL);
	
	// initialise neural networks with uniform distribution
    hiprandGenerateUniform(prng, dnns, population_size*parameter_size);

    // create random number generator for integer values
    unsigned int *random_int;
    hipMalloc((void**) &random_int,population_size*parameter_size*sizeof(int));

	hiprandGenerate(prng,random_int,population_size*parameter_size); 
    
    // initialse the neural networks to have negative values also
    initialise_nn<<<population_size,parameter_size>>>(dnns,random_int);

    // myprint<<<1,1>>>(dnns,population_size*parameter_size);

    // hipDeviceSynchronize();

    int *dfitness, *dindices;

	// fitness score on host
	int *fitness = (int *) malloc(population_size*sizeof(int));

	// fitness score on device
	hipMalloc((void**) &dfitness,population_size*sizeof(int));
	hipMalloc((void**) &dindices,population_size*sizeof(int));

    thrust::device_ptr<int> fitness_ptr(dfitness);
    thrust::device_ptr<int> indices_ptr(dindices);

    unsigned int *random_int_fruitx;
    hipMalloc((void**) &random_int_fruitx,population_size*max_snake_length*sizeof(int));
    unsigned int *random_int_fruity;
    hipMalloc((void**) &random_int_fruity,population_size*max_snake_length*sizeof(int));

    unsigned int *random_int_crossover1;
    hipMalloc((void**) &random_int_crossover1,2*population_size*sizeof(int));
    unsigned int *random_int_crossover2;
    hipMalloc((void**) &random_int_crossover2,population_size*sizeof(int));

    float *random_float_mutate1;
    hipMalloc((void**) &random_float_mutate1,population_size*parameter_size*sizeof(float));
    float *random_float_mutate2;
    hipMalloc((void**) &random_float_mutate2,population_size*parameter_size*sizeof(float));
    
    int max_reward = 0;
    float avg_reward = 0;
    int max_index = 0;
    int global_max_reward = 0;
    int global_max_generation = 0;

    float *best_snake = (float *)malloc(parameter_size*sizeof(float));

    hipStream_t stream1;
    hipStreamCreate(&stream1);
    for(int k=0;k<generations;k++){
        // cout<<"Generation: "<<k+1<<endl;

        // intialise indices array corresponding to fitness array
        int num_threads = (population_size > 1024) ? 1024 : population_size;
        int num_blocks = population_size/1024 + 1;
        intialise_indices<<<num_blocks,num_threads>>>(dindices);

        hiprandSetStream(prng,stream1);

        // create random number generator for integer values of fruit
        hiprandGenerate(prng,random_int_fruitx,parameter_size);
        hiprandGenerate(prng,random_int_fruity,parameter_size);
        
        play_game<<<population_size,parameter_size,parameter_size*sizeof(float)>>>(dnns,dfitness,random_int_fruitx,random_int_fruity,parameter_size);
        
        hipMemcpy(fitness,dfitness,population_size*sizeof(int),hipMemcpyDeviceToHost);
        
        avg_reward = 0;
        max_reward = fitness[0];
        max_index = 0;
        for(int i=1;i<population_size;i++){
            if(fitness[i] > max_reward){
                max_reward = fitness[i];
                max_index = i;
            }
            avg_reward += fitness[i];
        }
        avg_reward /= population_size;
        
        printf("generation: %d\tAverage fitness: %f\tMax reward: %d\n",k+1,avg_reward,max_reward);

        if(max_reward > global_max_reward){
            global_max_reward = max_reward;
            global_max_generation = k+1;
        }

        hipMemcpy(best_snake,dnns+max_index*parameter_size,parameter_size*sizeof(float),hipMemcpyDeviceToHost);

        hipDeviceSynchronize();

        if(k > 25)
            view_game(best_snake);
        
        int top = population_size * natural_selection_rate;

        thrust::sort_by_key(fitness_ptr,fitness_ptr+population_size,indices_ptr,thrust::greater<int>());

        // myprint1<<<1,1>>>(dfitness,dindices);

        select_top<<<top,parameter_size>>>(dnns,dnns_new,dindices);

        float *temp = dnns_new;
        dnns_new = dnns;
        dnns = temp;

        hiprandGenerate(prng,random_int_crossover1,2*population_size);
        hiprandGenerate(prng,random_int_crossover2,population_size);

        hiprandGenerateUniform(prng,random_float_mutate1,population_size*parameter_size);
        hiprandGenerateNormal(prng,random_float_mutate2,population_size*parameter_size,0.0,1.0);

        hipStreamSynchronize(stream1);
        
        crossover<<<population_size-top,parameter_size>>>(dnns,random_int_crossover1,random_int_crossover2);

        mutate<<<population_size,parameter_size>>>(dnns,random_float_mutate1,random_float_mutate2);

        // myprint<<<1,1>>>(dnns,parameter_size);

        // hipDeviceSynchronize();
    }

    printf("Generation: %d\tGlobal max reward: %d\n",global_max_generation,global_max_reward);

    hipStreamDestroy(stream1);
    hipFree(dnns);
    hipFree(dnns_new);
    hipFree(random_int);
    hipFree(dfitness);
    hipFree(dindices);
    hipFree(random_int_fruitx);
    hipFree(random_int_fruity);
    hipFree(random_int_crossover1);
    hipFree(random_int_crossover2);
    hipFree(random_float_mutate1);
    hipFree(random_float_mutate2);
    free(fitness);
    free(best_snake);

    return 0;
}